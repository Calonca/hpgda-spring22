#include "hip/hip_runtime.h"
//
// Created by calonca on 6/8/22.
//

#include "../personalized_pagerank.cuh"
#include "fastImplementation.cuh"

//////////////////////////////
//////////////////////////////

__global__ void incrementByValue(float *arr, int idx, float value) {
    arr[idx] += value;
}


__global__ void vectorScalarMul(const float scalar, float *vector, int array_len) {

    size_t tid = threadIdx.x,
            gridSize = blockDim.x * gridDim.x,
            i = blockIdx.x * blockDim.x + tid;

    while (i < array_len) {
        vector[i] *= scalar;
        i += gridSize;
    }
}
__global__ void vectorScalarAdd(float scalar, float *vector, int array_len) {

    size_t tid = threadIdx.x,
            gridSize = blockDim.x * gridDim.x,
            i = blockIdx.x * blockDim.x + tid;

    while (i < array_len) {
        vector[i] += scalar;
        i += gridSize;
    }
}

__global__ void init_vector(float* v, int size, float value) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < size) {
        v[i] = value;
        i += gridDim.x * blockDim.x;
    }
}

__global__ void copy_vector(float* dest, int* source, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < size) {
        dest[i] = source[i];
        i += gridDim.x * blockDim.x;
    }
}

//////////////////////////////
//////////////////////////////


void FastImplementation::dotCublas(const float *array1, const float *array2, float *result, int vector_len, float alpha,
                                   hipblasHandle_t *handle) {

    hipblasSdot (*handle, vector_len,
                         array1, 1,
                         array2, 1,
                         result);

    *result=alpha*(*result)/vector_len;
}

void FastImplementation::sqCublas(const float *old, float *newVector, float *result, int vector_len, hipblasHandle_t *handle) {

    float A = -1.0;
    const float* pA = &A;

    hipblasSaxpy(*handle, vector_len,
                         pA,
                         old, 1,
                         newVector, 1);
    hipDeviceSynchronize();


    hipblasSdot (*handle, vector_len,
                         newVector, 1,
                         newVector, 1,
                         result);

}


void FastImplementation::alloc() {
    // Load the input graph and preprocess it;
    pPpr->initialize_graph();

    hipMallocManaged(&coo.x, sizeof(int) * pPpr->E);
    hipMallocManaged(&csr.rowIndex, sizeof(int) * pPpr->E);
    hipMallocManaged(&coo.y, sizeof(int) * pPpr->E);
    hipMallocManaged(&coo.val, sizeof(float ) * pPpr->E);


    hipMallocManaged(&dangling_gpu, sizeof(int) * pPpr->V);
    hipMallocManaged(&pSquareError_gpu, sizeof(float ));

    hipMallocManaged(&pr_gpu, sizeof(float)*pPpr->V);
    hipMallocManaged(&pr_temp, sizeof(float)*pPpr->V);
    hipMallocManaged(&pr_old, sizeof(float)*pPpr->V);

    hipMallocManaged(&pDanglingFact_gpu, sizeof(float ));
    hipMallocManaged(&pAlpha_gpu, sizeof(float ));
    hipMallocManaged(&pTeleportFact_gpu, sizeof(float ));
    hipMallocManaged(&pV_gpu, sizeof(int ));
    hipMallocManaged(&pE_gpu, sizeof(int ));
    hipMallocManaged(&pPersonalization_vertex_gpu, sizeof(int ));
}

//Print error if the block size is too small
void FastImplementation::checkBlockSize(int blockSize, int minBlockSize) {
    if (blockSize < minBlockSize && debug) {
        printf("The selected number of blocks is too small, %d will be used\n",minBlockSize);
    }
}

void FastImplementation::checkCuSparseStatus(hipsparseStatus_t status) {
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        std::string error_string = hipsparseGetErrorString(status);
        printf("Cusparse error: %s\n", error_string.c_str());
    }
}


void FastImplementation::init() {
    hipblasCreate(&cublasHandle);

    hipsparseCreate(&cusparseHandle);

    valFloat.resize(pPpr->E);
    std::transform(pPpr->val.begin(), pPpr->val.end(), valFloat.begin(), [](double x) { return (float )x;});

    blocksVertex = 1+pPpr->V/pPpr->T;
    blocksEdge = 1+pPpr->E/pPpr->T;

    int blocks = std::max(blocksVertex,blocksEdge);

    checkBlockSize(pPpr->B,blocks);

    hipMemcpy(coo.x,pPpr->x.data(), sizeof(int) * pPpr->E, hipMemcpyHostToDevice);
    hipMemcpy(coo.y,pPpr->y.data(), sizeof(int) * pPpr->E,hipMemcpyHostToDevice);
    hipMemcpy(coo.val,valFloat.data(), sizeof(float ) * pPpr->E,hipMemcpyHostToDevice);
    vectorScalarMul<<<blocksEdge, pPpr->T>>>(pPpr->alpha, coo.val, pPpr->E);
    hipMemcpy(dangling_gpu, pPpr->dangling.data(), sizeof(int) * pPpr->V, hipMemcpyHostToDevice);

    initCoo();
}

void FastImplementation::reset() {
    hipMemset(pSquareError_gpu,INITIAL_SQUARE_ERROR, sizeof(float ));

    hipMemset(pr_gpu,0.0, sizeof(float)*pPpr->V);
    hipMemset(pr_temp,0.0, sizeof(float)*pPpr->V);
    hipMemset(pr_old,0.0, sizeof(float)*pPpr->V);

    hipMemset(pDanglingFact_gpu,0, sizeof(float ));
    hipMemcpy(pAlpha_gpu,&(pPpr->alpha), sizeof(float ),hipMemcpyHostToDevice);
    float tempTeleportFact = pPpr->alpha/pPpr->V;
    hipMemcpy(pTeleportFact_gpu,&tempTeleportFact, sizeof(float ),hipMemcpyHostToDevice);

    hipMemcpy(pV_gpu,&(pPpr->V), sizeof(int ),hipMemcpyHostToDevice);
    hipMemcpy(pE_gpu,&(pPpr->E), sizeof(int ),hipMemcpyHostToDevice);
    hipMemcpy(pPersonalization_vertex_gpu,&(pPpr->personalization_vertex), sizeof(int ),hipMemcpyHostToDevice);

    init_vector<<<blocksVertex,pPpr->T>>>(pr_old,pPpr->V,1.0f/float(pPpr->V));

}

void FastImplementation::execute(int iter) {

    if (debug) {
        printf("\nx initial value\n");
        print_gpu_array(coo.x, pPpr->E);
        printf("\ny initial value\n");
        print_gpu_array(coo.y, pPpr->E);
        printf("\nval initial value\n");
        print_gpu_array(coo.val, pPpr->E);
        printf("\npr_old initial value\n");
        print_gpu_array(pr_old, pPpr->V);
    }

    float squareError_cpu = INITIAL_SQUARE_ERROR;

    hipMemset(pDanglingFact_gpu,0, sizeof(float ));
    hipMemset(pr_temp,0, sizeof(float )*pPpr->V);
    hipMemset(pSquareError_gpu,0, sizeof(float ));

    for (int i = 0; squareError_cpu > pPpr->convergence_threshold && i < pPpr->max_iterations; i++) {
        if (debug){
            printf("\nConvergence iteration %i pr_old\n",i);
            print_gpu_array(pr_old,pPpr->V);
            printf("\nSum is\n");
            printGpu_vector_sum(pr_old,pPpr->V);}

        hipMemset(pDanglingFact_gpu,0, sizeof(float ));
        hipMemset(pr_temp,0, sizeof(float )*pPpr->V);
        hipMemset(pSquareError_gpu,0, sizeof(float ));

        thrust::device_vector<float> danglingFloat(pPpr->V);
        copy_vector<<<blocksVertex,pPpr->T>>>(thrust::raw_pointer_cast(danglingFloat.data()),dangling_gpu,pPpr->V);
        dotCublas(thrust::raw_pointer_cast(danglingFloat.data()), pr_old, pDanglingFact_gpu, *pV_gpu, pPpr->alpha,
                  &cublasHandle);
        hipDeviceSynchronize();

        if (debug)
            printf("\nIteration %i dangling is: %f \n",i,*pDanglingFact_gpu);


        float alpha = 1.0f;
        float beta = 0.0f;
        size_t bufferSize = pPpr->V * sizeof(float);

        checkCuSparseStatus(hipsparseDnVecSetValues(pr_old_descr,pr_old));
        //hipsparseDnVecGetValues(pr_old_descr, reinterpret_cast<void **>(&pr_temp));
        hipDeviceSynchronize();


        checkCuSparseStatus(hipsparseSpMV_bufferSize(cusparseHandle,
                                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                    &alpha,
                                                    coo.descr,
                                                    pr_old_descr,
                                                    &beta,
                                                    pr_temp_descr,
                                                    HIP_R_32F,
                                                    coo.alg,
                                                    &bufferSize));

        hipDeviceSynchronize();

        void *buffer;
        if (bufferSize>0) {
            hipMalloc(&buffer, bufferSize);
            std::cout << "\nBuffer size is: " << bufferSize << std::endl;
        }

        checkCuSparseStatus(hipsparseSpMV(cusparseHandle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         coo.descr,
                                         pr_old_descr,
                                         &beta,
                                         pr_temp_descr,
                                         HIP_R_32F,
                                         coo.alg,
                                         &buffer));

        if (bufferSize>0)
            hipFree(buffer);
        checkCuSparseStatus(hipsparseDnVecGetValues(pr_temp_descr, reinterpret_cast<void **>(&pr_temp)));
        hipDeviceSynchronize();

        if (debug) {
            printf("\npr_temp after coo\n");
            print_gpu_array(pr_temp, pPpr->V);
            printf("\nSum is\n");
            printGpu_vector_sum(pr_temp,pPpr->V);
        }

        vectorScalarAdd<<<blocksVertex,pPpr->T>>>(*pDanglingFact_gpu,pr_temp,*pV_gpu);
        hipDeviceSynchronize();

        if(debug) {
            printf("\npr_temp after adding dangling\n");
            print_gpu_array(pr_temp, pPpr->V);
            printf("\nSum is\n");
            printGpu_vector_sum(pr_temp, pPpr->V);
        }

        incrementByValue<<<1, 1>>>(pr_temp, pPpr->personalization_vertex, 1.0 - pPpr->alpha);

        hipMemcpy(pr_gpu,pr_temp ,sizeof(float)*pPpr->V,hipMemcpyDeviceToDevice);

        hipDeviceSynchronize();

        if (debug) {
            printf("\npr_gpu after increment of 1-alpha in position %d\n", pPpr->personalization_vertex);
            print_gpu_array(pr_gpu, 20);
            printf("\nSum is\n");
            printGpu_vector_sum(pr_gpu, pPpr->V);
        }

        sqCublas(pr_old, pr_temp,pSquareError_gpu, pPpr->V,&cublasHandle);

        hipMemcpy(&squareError_cpu,pSquareError_gpu,sizeof (float ),hipMemcpyDeviceToHost);
        squareError_cpu= std::sqrt(squareError_cpu);
        hipDeviceSynchronize();

        if (debug)
        {
            printf("\nIteration %i Square error is: %f \n",i,squareError_cpu);
        }
        hipMemcpy(pr_old,pr_gpu ,sizeof(float)*pPpr->V,hipMemcpyDeviceToDevice);
    }

    prFloat.resize(pPpr->V);
    hipMemcpy(prFloat.data(),pr_gpu,sizeof (float )*pPpr->V,hipMemcpyDeviceToHost);
    std::transform(prFloat.begin(), prFloat.end(), pPpr->pr.begin(), [](float x) { return (double )x;});
}

void FastImplementation::clean() {
    hipFree(coo.x);
    hipFree(coo.y);
    hipFree(coo.val);

    hipFree(dangling_gpu);
    hipFree(pSquareError_gpu);

    hipFree(pr_gpu);
    hipFree(pr_temp);
    hipFree(pr_old);


    hipFree(pDanglingFact_gpu);
    hipFree(pAlpha_gpu);
    hipFree(pTeleportFact_gpu);
    hipFree(pV_gpu);
    hipFree(pE_gpu);
    hipFree(pPersonalization_vertex_gpu);


    hipblasDestroy(cublasHandle);
    hipsparseDestroy(cusparseHandle);

}

void FastImplementation::initCoo() {
    checkCuSparseStatus(hipsparseCreateDnVec(&pr_temp_descr,pPpr->V,pr_temp,HIP_R_32F));
    checkCuSparseStatus(hipsparseCreateDnVec(&pr_old_descr,pPpr->V,pr_old,HIP_R_32F));


    checkCuSparseStatus(hipsparseCreateCoo(
            &coo.descr,
            pPpr->V,
            pPpr->V,
            pPpr->E,
            coo.x,
            coo.y,
            coo.val,
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO,
            HIP_R_32F));

}

