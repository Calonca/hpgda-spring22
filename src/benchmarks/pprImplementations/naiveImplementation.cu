#include "hip/hip_runtime.h"
#include "naiveImplementation.cuh"
#include "../personalized_pagerank.cuh"
#include "../pprFunctions/pprFunctions.cuh"

void NaiveImplementation::alloc() {
    // Load the input graph and preprocess it;
    pPpr->initialize_graph();

    hipMallocManaged(&x_gpu, sizeof(int) * pPpr->E);
    hipMallocManaged(&y_gpu, sizeof(int) * pPpr->E);
    hipMallocManaged(&val_gpu, sizeof(double ) * pPpr->E);


    hipMallocManaged(&dangling_gpu, sizeof(int) * pPpr->V);
    hipMallocManaged(&pSquareError_gpu, sizeof(double ));

    hipMallocManaged(&pr_gpu, sizeof(double)*pPpr->V);
    hipMallocManaged(&pr_temp, sizeof(double)*pPpr->V);
    hipMallocManaged(&pr_old, sizeof(double)*pPpr->V);

    hipMallocManaged(&pDanglingFact_gpu, sizeof(double ));
    hipMallocManaged(&pAlpha_gpu, sizeof(double ));
    hipMallocManaged(&pTeleportFact_gpu, sizeof(double ));
    hipMallocManaged(&pV_gpu, sizeof(int ));
    hipMallocManaged(&pE_gpu, sizeof(int ));
    hipMallocManaged(&pPersonalization_vertex_gpu, sizeof(int ));

    BLOCKS_V = (pPpr->V + THREADS - 1)/ THREADS;
    BLOCKS_E = (pPpr->E + THREADS - 1)/ THREADS;
}

void NaiveImplementation::init() {
}

void NaiveImplementation::reset() {

    hipMemcpy(x_gpu,pPpr->x.data(), sizeof(int) * pPpr->E, hipMemcpyHostToDevice);

    hipMemcpy(y_gpu,pPpr->y.data(), sizeof(int) * pPpr->E,hipMemcpyHostToDevice);
    hipMemcpy(val_gpu,pPpr->val.data(), sizeof(double ) * pPpr->E,hipMemcpyHostToDevice);
    vectorScalarMul<double><<<BLOCKS_E, THREADS>>>(pPpr->alpha, val_gpu, pPpr->E);

    hipMemcpy(dangling_gpu, pPpr->dangling.data(), sizeof(int) * pPpr->V, hipMemcpyHostToDevice);
    hipMemset(pr_gpu,0.0, sizeof(double)*pPpr->V);
    hipMemset(pr_temp,0.0, sizeof(double)*pPpr->V);
    hipMemcpy(pr_old,pPpr->pr.data() ,sizeof(double)*pPpr->V,hipMemcpyHostToDevice);

    hipMemset(pDanglingFact_gpu,0.0, sizeof(double ));
    hipMemcpy(pAlpha_gpu,&(pPpr->alpha), sizeof(double ),hipMemcpyHostToDevice);
    double tempTeleportFact = pPpr->alpha/pPpr->V;
    hipMemcpy(pTeleportFact_gpu,&tempTeleportFact, sizeof(double ),hipMemcpyHostToDevice);

    hipMemcpy(pV_gpu,&(pPpr->V), sizeof(int ),hipMemcpyHostToDevice);
    hipMemcpy(pE_gpu,&(pPpr->E), sizeof(int ),hipMemcpyHostToDevice);
    hipMemcpy(pPersonalization_vertex_gpu,&(pPpr->personalization_vertex), sizeof(int ),hipMemcpyHostToDevice);
}

void NaiveImplementation::execute(int iter) {

    double squareError_cpu = INITIAL_SQUARE_ERROR;

    hipMemset(pDanglingFact_gpu,0.0, sizeof(double ));
    hipMemset(pr_temp,0.0, sizeof(double )*pPpr->V);
    hipMemset(pSquareError_gpu,0.0, sizeof(double ));

    for (int i = 0; squareError_cpu > pPpr->convergence_threshold && i < pPpr->max_iterations; i++) {

        hipMemset(pDanglingFact_gpu,0.0, sizeof(double));
        init_vector<double><<<BLOCKS_V, THREADS>>>(pr_temp,pPpr->V, 0.0);
        hipMemset(pSquareError_gpu,0.0, sizeof(double ));
        hipDeviceSynchronize();

        compute_dangling_factor_gpu<int, double><<<BLOCKS_V, THREADS>>>(dangling_gpu, pr_old, pDanglingFact_gpu, *pV_gpu);
        hipDeviceSynchronize();

        cooSPMV<int, double><<<BLOCKS_E,THREADS>>>(x_gpu,y_gpu,val_gpu,pPpr->E,pr_old,pr_temp);
        hipDeviceSynchronize();

        vectorScalarAdd<double><<<BLOCKS_V,THREADS>>>(pDanglingFact_gpu[0] * (pPpr->alpha / pPpr->V),pr_temp,*pV_gpu);
        hipDeviceSynchronize();

        copy_vector<double><<<BLOCKS_V, THREADS>>>(pr_gpu,pr_temp, pPpr->V);
        hipDeviceSynchronize();

        incremetBy1<double><<<1, 1>>>(pr_gpu, pPpr->personalization_vertex,1.0-pPpr->alpha);
        hipDeviceSynchronize();

        compute_square_error_gpu<double><<<BLOCKS_V, THREADS>>>(pr_old, pr_gpu, pSquareError_gpu, pPpr->V);
        hipDeviceSynchronize();

        hipMemcpy(&squareError_cpu,pSquareError_gpu,sizeof (double ),hipMemcpyDeviceToHost);

        squareError_cpu= std::sqrt(squareError_cpu);

        copy_vector<double><<<BLOCKS_V, THREADS>>>(pr_old,pr_gpu, pPpr->V);
        hipDeviceSynchronize();

    }

    hipDeviceSynchronize();
    //A pointer to the address in base class is used since the validation is done by the base class
    hipMemcpy(pPpr->pr.data(),pr_gpu,sizeof(double)*pPpr->V,hipMemcpyDeviceToHost);
}

void NaiveImplementation::clean() {
    hipFree(x_gpu);
    hipFree(y_gpu);
    hipFree(val_gpu);
    hipFree(dangling_gpu);
    hipFree(pSquareError_gpu);
    hipFree(pr_gpu);
    hipFree(pr_temp);
    hipFree(pr_old);
    hipFree(pDanglingFact_gpu);
    hipFree(pAlpha_gpu);
    hipFree(pTeleportFact_gpu);
    hipFree(pV_gpu);
    hipFree(pE_gpu);
    hipFree(pPersonalization_vertex_gpu);
}
