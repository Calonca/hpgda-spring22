#include "hip/hip_runtime.h"
//
// Created by calonca on 6/8/22.
//

#include "../personalized_pagerank.cuh"
#include "cublasCusparseNaiveImplementation.cuh"

//////////////////////////////
//////////////////////////////

__global__ void incrementByValue_n(float *arr, int idx, float value) {
    arr[idx] += value;
}


__global__ void vectorScalarMul_n(const float scalar, float *vector, int array_len) {

    size_t tid = threadIdx.x,
            gridSize = blockDim.x * gridDim.x,
            i = blockIdx.x * blockDim.x + tid;

    while (i < array_len) {
        vector[i] *= scalar;
        i += gridSize;
    }
}
__global__ void vectorScalarAdd_n(float scalar, float *vector, int array_len) {

    size_t tid = threadIdx.x,
            gridSize = blockDim.x * gridDim.x,
            i = blockIdx.x * blockDim.x + tid;

    while (i < array_len) {
        vector[i] += scalar;
        i += gridSize;
    }
}

__global__ void init_vector_n(float* v, int size, float value) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < size) {
        v[i] = value;
        i += gridDim.x * blockDim.x;
    }
}

__global__ void copy_vector_n(float* dest, int* source, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < size) {
        dest[i] = source[i];
        i += gridDim.x * blockDim.x;
    }
}

//////////////////////////////
//////////////////////////////


void CublasCusparseNaiveImplementation::dotCublas(const float *array1, const float *array2, float *result, int vector_len, float alpha, hipblasHandle_t* handle){

    hipblasSdot (*handle, vector_len,
                         array1, 1,
                         array2, 1,
                         result);

    *result=alpha*(*result)/vector_len;
}

void CublasCusparseNaiveImplementation::sqCublas(const float *old,float *newVector, float* result, int vector_len,hipblasHandle_t* handle){

    float A = -1.0;
    const float* pA = &A;

    hipblasSaxpy(*handle, vector_len,
                         pA,
                         old, 1,
                         newVector, 1);
    hipDeviceSynchronize();

    //if (status!= HIPBLAS_STATUS_SUCCESS){
    //    printf("Error cudblast %d\n",static_cast<int>(status));
    //}

    hipblasSdot (*handle, vector_len,
                         newVector, 1,
                         newVector, 1,
                         result);

    //if (status!= HIPBLAS_STATUS_SUCCESS){
    //    printf("Error cudblast %d\n",static_cast<int>(status));
    //}
}


void CublasCusparseNaiveImplementation::alloc() {
    // Load the input graph and preprocess it;
    pPpr->initialize_graph();
    bsr.blocksInMat = (pPpr->V + bsr.bsrBlockDim - 1) / bsr.bsrBlockDim;//Number of blocks in the matrix, composed by blocksInMat*bsrBlockDim blocks

    hipMallocManaged(&coo.x_gpu, sizeof(int) * pPpr->E);
    hipMallocManaged(&csr.rowIndex, sizeof(int) * pPpr->E);
    hipMallocManaged(&coo.y_gpu, sizeof(int) * pPpr->E);
    hipMallocManaged(&coo.val_gpu, sizeof(float ) * pPpr->E);


    hipMallocManaged(&dangling_gpu, sizeof(int) * pPpr->V);
    hipMallocManaged(&pSquareError_gpu, sizeof(float ));

    hipMallocManaged(&pr_gpu, sizeof(float)*pPpr->V);
    hipMallocManaged(&pr_temp, sizeof(float)*pPpr->V+bsr.bsrBlockDim);
    hipMallocManaged(&pr_old, sizeof(float)*pPpr->V+bsr.bsrBlockDim);

    hipMallocManaged(&pDanglingFact_gpu, sizeof(float ));
    hipMallocManaged(&pAlpha_gpu, sizeof(float ));
    hipMallocManaged(&pTeleportFact_gpu, sizeof(float ));
    hipMallocManaged(&pV_gpu, sizeof(int ));
    hipMallocManaged(&pE_gpu, sizeof(int ));
    hipMallocManaged(&pPersonalization_vertex_gpu, sizeof(int ));
}

//Print error if the block size is too small
void CublasCusparseNaiveImplementation::checkBlockSize(int blockSize,int minBlockSize) {
    if (blockSize < minBlockSize && debug) {
        printf("The selected number of blocks is too small, %d will be used\n",minBlockSize);
    }
}


void CublasCusparseNaiveImplementation::checkCuSparseStatus(hipsparseStatus_t status) {
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        std::string error_string = hipsparseGetErrorString(status);
        printf("Cusparse error: %s\n", error_string.c_str());
    }
}


void CublasCusparseNaiveImplementation::init() {
    hipblasCreate(&cublasHandle);

    hipsparseCreate(&cusparseHandle);

    valFloat.resize(pPpr->E);
    std::transform(pPpr->val.begin(), pPpr->val.end(), valFloat.begin(), [](double x) { return (float )x;});

    blocksVertex = 1+pPpr->V/pPpr->T;
    blocksEdge = 1+pPpr->E/pPpr->T;

    int blocks = std::max(blocksVertex,blocksEdge);

    checkBlockSize(pPpr->B,blocks);

    hipMemset(pr_gpu,0.0, sizeof(float)*pPpr->V);
    hipMemset(pr_temp,0.0, sizeof(float)*pPpr->V+bsr.bsrBlockDim);

    hipMemcpy(coo.x_gpu,pPpr->x.data(), sizeof(int) * pPpr->E, hipMemcpyHostToDevice);
    hipMemcpy(coo.y_gpu,pPpr->y.data(), sizeof(int) * pPpr->E,hipMemcpyHostToDevice);
    hipMemcpy(coo.val_gpu,valFloat.data(), sizeof(float ) * pPpr->E,hipMemcpyHostToDevice);
    vectorScalarMul_n<<<blocksEdge, pPpr->T>>>(pPpr->alpha, coo.val_gpu, pPpr->E);
    hipDeviceSynchronize();

    hipMallocManaged((void**)&bsr.rowPtrs, sizeof(int) * (bsr.blocksInMat + 1));
    cooToBsr();
}

void CublasCusparseNaiveImplementation::reset() {
    hipMemcpy(coo.x_gpu,pPpr->x.data(), sizeof(int) * pPpr->E, hipMemcpyHostToDevice);

    hipMemcpy(coo.y_gpu,pPpr->y.data(), sizeof(int) * pPpr->E,hipMemcpyHostToDevice);

    hipMemcpy(coo.val_gpu,valFloat.data(), sizeof(float ) * pPpr->E,hipMemcpyHostToDevice);
    vectorScalarMul_n<<<blocksEdge, pPpr->T>>>(pPpr->alpha, coo.val_gpu, pPpr->E);

    hipMemcpy(dangling_gpu, pPpr->dangling.data(), sizeof(int) * pPpr->V, hipMemcpyHostToDevice);

    hipMemset(pSquareError_gpu,INITIAL_SQUARE_ERROR, sizeof(float ));

    hipMemset(pr_gpu,0.0, sizeof(float)*pPpr->V);
    hipMemset(pr_temp,0.0, sizeof(float)*bsr.blocksInMat*bsr.bsrBlockDim);
    hipMemset(pr_old,0.0, sizeof(float)*bsr.blocksInMat*bsr.bsrBlockDim);

    hipMemset(pDanglingFact_gpu,0, sizeof(float ));
    hipMemcpy(pAlpha_gpu,&(pPpr->alpha), sizeof(float ),hipMemcpyHostToDevice);
    float tempTeleportFact = pPpr->alpha/pPpr->V;
    hipMemcpy(pTeleportFact_gpu,&tempTeleportFact, sizeof(float ),hipMemcpyHostToDevice);

    hipMemcpy(pV_gpu,&(pPpr->V), sizeof(int ),hipMemcpyHostToDevice);
    hipMemcpy(pE_gpu,&(pPpr->E), sizeof(int ),hipMemcpyHostToDevice);
    hipMemcpy(pPersonalization_vertex_gpu,&(pPpr->personalization_vertex), sizeof(int ),hipMemcpyHostToDevice);

    init_vector_n<<<blocksVertex,pPpr->T>>>(pr_old,pPpr->V,1.0f/float(pPpr->V));

}

void CublasCusparseNaiveImplementation::execute(int iter) {

    if (debug) {
        printf("\nx_gpu initial value\n");
        print_gpu_array(coo.x_gpu, pPpr->E);
        printf("\ny_gpu initial value\n");
        print_gpu_array(coo.y_gpu, pPpr->E);
        printf("\nval_gpu initial value\n");
        print_gpu_array(coo.val_gpu, pPpr->E);
        printf("\npr_old initial value\n");
        print_gpu_array(pr_old, pPpr->V);
    }

    float squareError_cpu = INITIAL_SQUARE_ERROR;

    hipMemset(pDanglingFact_gpu,0, sizeof(float ));
    hipMemset(pr_temp,0, sizeof(float )*bsr.blocksInMat*bsr.bsrBlockDim);
    hipMemset(pSquareError_gpu,0, sizeof(float ));

    for (int i = 0; squareError_cpu > pPpr->convergence_threshold && i < pPpr->max_iterations; i++) {
        if (debug){
            printf("\nConvergence iteration %i pr_old\n",i);
            print_gpu_array(pr_old,pPpr->V);
            printf("\nSum is\n");
            printGpu_vector_sum(pr_old,pPpr->V);}

        hipMemset(pDanglingFact_gpu,0, sizeof(float ));
        hipMemset(pr_temp,0, sizeof(float )*bsr.blocksInMat*bsr.bsrBlockDim);
        hipMemset(pSquareError_gpu,0, sizeof(float ));

        thrust::device_vector<float> danglingFloat(pPpr->V);
        copy_vector_n<<<blocksVertex,pPpr->T>>>(thrust::raw_pointer_cast(danglingFloat.data()),dangling_gpu,pPpr->V);
        dotCublas(thrust::raw_pointer_cast(danglingFloat.data()), pr_old, pDanglingFact_gpu, *pV_gpu, pPpr->alpha,
                  &cublasHandle);
        hipDeviceSynchronize();

        if (debug)
            printf("\nIteration %i dangling is: %f \n",i,*pDanglingFact_gpu);


        float alpha = 1.0f;
        float beta = 0.0f;
        //size_t bufferSize = pPpr->V * sizeof(float);
        /*
        checkCuSparseStatus(hipsparseDnVecSetValues(pr_old_descr,pr_old));
        //hipsparseDnVecGetValues(pr_old_descr, reinterpret_cast<void **>(&pr_temp));
        hipDeviceSynchronize();


        checkCuSparseStatus(hipsparseSpMV_bufferSize(cusparseHandle,
                                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                    &alpha,
                                                    coo.descr,
                                                    pr_old_descr,
                                                    &beta,
                                                    pr_temp_descr,
                                                    HIP_R_32F,
                                                    coo.alg,
                                                    &bufferSize));

        hipDeviceSynchronize();

        void *buffer;
        if (bufferSize>0) {//Naive way to create a buffer for coo alg
            hipMalloc(&buffer, bufferSize);
            std::cout << "\nBuffer size is: " << bufferSize << std::endl;
        }

        checkCuSparseStatus(hipsparseSpMV(cusparseHandle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         coo.descr,
                                         pr_old_descr,
                                         &beta,
                                         pr_temp_descr,
                                         HIP_R_32F,
                                         coo.alg,
                                         &buffer));

        if (bufferSize>0)
            hipFree(buffer);
        checkCuSparseStatus(hipsparseDnVecGetValues(pr_temp_descr, reinterpret_cast<void **>(&pr_temp)));

        */

        checkCuSparseStatus(hipsparseSbsrmv(
                cusparseHandle, bsr.dir, HIPSPARSE_OPERATION_NON_TRANSPOSE, bsr.blocksInMat, bsr.blocksInMat, bsr.nnZBlocks,
                &alpha,
                bsr.bsr_desc,
                bsr.val, bsr.rowPtrs,bsr.colIdxs, bsr.bsrBlockDim,
                pr_old, &beta,pr_temp ));


        hipDeviceSynchronize();

        hipDeviceSynchronize();

        if (debug) {
            printf("\npr_temp after coo\n");
            print_gpu_array(pr_temp, pPpr->V);
            printf("\nSum is\n");
            printGpu_vector_sum(pr_temp,pPpr->V);
        }

        vectorScalarAdd_n<<<blocksVertex,pPpr->T>>>(*pDanglingFact_gpu,pr_temp,*pV_gpu);
        hipDeviceSynchronize();

        if(debug) {
            printf("\npr_temp after adding dangling\n");
            print_gpu_array(pr_temp, pPpr->V);
            printf("\nSum is\n");
            printGpu_vector_sum(pr_temp, pPpr->V);
        }

        //if (noPersonalizedVertex) {
        //    vectorScalarAdd_n<<<blocksVertex,pPpr->T>>>((1.0 - pPpr->alpha)/pPpr->V,pr_temp,*pV_gpu);
        //}else {
            incrementByValue_n<<<1, 1>>>(pr_temp, pPpr->personalization_vertex, 1.0 - pPpr->alpha);
        //}
        hipMemcpy(pr_gpu,pr_temp ,sizeof(float)*pPpr->V,hipMemcpyDeviceToDevice);

        hipDeviceSynchronize();

        if (debug) {
            printf("\npr_gpu after increment of 1-alpha in position %d\n", pPpr->personalization_vertex);
            print_gpu_array(pr_gpu, 20);
            printf("\nSum is\n");
            printGpu_vector_sum(pr_gpu, pPpr->V);
        }

        //squareError
        //compute_square_error_gpu<<<blocksVertex, pPpr->T>>>(pr_old, pr_gpu,pSquareError_gpu, pPpr->V);
        //pr_temp will be overwritten
        sqCublas(pr_old, pr_temp,pSquareError_gpu, pPpr->V,&cublasHandle);

        hipMemcpy(&squareError_cpu,pSquareError_gpu,sizeof (float ),hipMemcpyDeviceToHost);
        squareError_cpu= std::sqrt(squareError_cpu);
        hipDeviceSynchronize();

        if (debug)
        {
            printf("\nIteration %i Square error is: %f \n",i,squareError_cpu);
        }
        hipMemcpy(pr_old,pr_gpu ,sizeof(float)*pPpr->V,hipMemcpyDeviceToDevice);
    }

    prFloat.resize(pPpr->V);
    hipMemcpy(prFloat.data(),pr_gpu,sizeof (float )*pPpr->V,hipMemcpyDeviceToHost);
    std::transform(prFloat.begin(), prFloat.end(), pPpr->pr.begin(), [](float x) { return (double )x;});
}

void CublasCusparseNaiveImplementation::clean() {
    hipFree(coo.x_gpu);
    hipFree(coo.y_gpu);
    hipFree(coo.val_gpu);

    hipFree(dangling_gpu);
    hipFree(pSquareError_gpu);

    hipFree(pr_gpu);
    hipFree(pr_temp);
    hipFree(pr_old);


    hipFree(pDanglingFact_gpu);
    hipFree(pAlpha_gpu);
    hipFree(pTeleportFact_gpu);
    hipFree(pV_gpu);
    hipFree(pE_gpu);
    hipFree(pPersonalization_vertex_gpu);


    hipblasDestroy(cublasHandle);
    hipsparseDestroy(cusparseHandle);

}

void CublasCusparseNaiveImplementation::cooToBsr() {
    checkCuSparseStatus(hipsparseCreateDnVec(&coo.x_descr,pPpr->V,coo.x_gpu,HIP_R_32F));
    checkCuSparseStatus(hipsparseCreateDnVec(&pr_temp_descr,pPpr->V,pr_temp,HIP_R_32F));
    checkCuSparseStatus(hipsparseCreateDnVec(&pr_old_descr,pPpr->V,pr_old,HIP_R_32F));


    checkCuSparseStatus(hipsparseCreateCoo(
            &coo.descr,
            pPpr->V,
            pPpr->V,
            pPpr->E,
            coo.x_gpu,
            coo.y_gpu,
            coo.val_gpu,
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO,
            HIP_R_32F));


    checkCuSparseStatus(hipsparseXcoo2csr(
            cusparseHandle,
            coo.x_gpu,pPpr->E,pPpr->V,
            csr.rowIndex,
            HIPSPARSE_INDEX_BASE_ZERO)
    );

    /*
    checkCuSparseStatus(hipsparseCreateCsr(
            reinterpret_cast<hipsparseSpMatDescr_t *>(&mat_desc),

            pPpr->V,pPpr->V,pPpr->E,
            rowIndex,y_gpu,val_gpu,
            HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO,
            HIP_R_32F));*/

    hipsparseCreateMatDescr(&csr.mat_desc);
    hipsparseSetMatType(csr.mat_desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csr.mat_desc, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseCreateMatDescr(&bsr.bsr_desc);
    hipsparseSetMatType(bsr.bsr_desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsr.bsr_desc, HIPSPARSE_INDEX_BASE_ZERO);

    //Computer number of nonZeroes blocks
    hipDeviceSynchronize();
    checkCuSparseStatus((hipsparseXcsr2bsrNnz(
            cusparseHandle, bsr.dir, pPpr->V, pPpr->V,
            csr.mat_desc, csr.rowIndex, coo.y_gpu, bsr.bsrBlockDim,
            bsr.bsr_desc, bsr.rowPtrs, &bsr.nnZBlocks)));
    hipMallocManaged((void**)&bsr.colIdxs, sizeof(int) * bsr.nnZBlocks);
    hipMallocManaged((void**)&bsr.val, sizeof(float) * (bsr.bsrBlockDim * bsr.bsrBlockDim) * bsr.nnZBlocks);
    hipDeviceSynchronize();
    //Convert from csr to bsr
    checkCuSparseStatus(hipsparseScsr2bsr(
            cusparseHandle, bsr.dir, pPpr->V, pPpr->V,
            csr.mat_desc, coo.val_gpu, csr.rowIndex, coo.y_gpu, bsr.bsrBlockDim,
            bsr.bsr_desc, bsr.val, bsr.rowPtrs, bsr.colIdxs));
    hipDeviceSynchronize();
    /*
    for(auto i = 0; i < min(20,(bsr.blocksInMat + 1)); i++) {
        printf("bsrRowPtrC[%2d] = %d\n", i, bsr.rowPtrs[i]);
    }
    printf("\n");

    for(auto i = 0; i < min(bsr.nnZBlocks,20); i++) {
        printf("bsrColIndC[%2d] = %d\n", i, bsr.colIdxs[i]);
    }
    printf("\n");

    for(auto i = 0; i < min(20,(bsr.bsrBlockDim * bsr.bsrBlockDim) * bsr.nnZBlocks); i++) {
        printf("bsrVal[%2d] = %f\n", i, bsr.val[i]);
    }*/
}





