#include "hip/hip_runtime.h"
#include "finalImplementation.cuh"
#include "../pprFunctions/pprFunctions.cuh"
#include "../pprFunctions/coospmv.cuh"
#include <algorithm>

void FinalImplementation::initDanglingPprTop19(){

    float val = 10.0;

    // California top19
    // int top19[19] = {16,40,65, 210, 996, 1078, 1082, 1487, 1488, 1616, 1805, 1861, 1862, 2077, 2407, 4390, 4822, 6426, 9663};

    // Wikipedia top19
    // 0000007030 0000009008 0000024716 0000027566 0000028020 0000028196 0000195101 0000500469 0000577659 0000689491 0000932394 0001518892 0001702309 0001835017 0002144742 0002257865 0002532493 0002979297 0002984189
    int top19[19] = {7030, 9008, 24716, 27566, 28020, 28196, 195101, 500469, 577659, 689491, 932394, 1518892, 1702309, 1835017, 2144742, 2257865, 2532493, 2979297, 2984189};

    for(int i=0; i<19; i++){
        danglingPprTop19.insert(std::pair<int, double>(top19[i], val));
        val = val-0.1;
    }

}

void FinalImplementation::initDanglingPpr(){
    float val;
    for(int i=0; i<pPpr->V; i++){
        if(danglingPprTop19.count(i)){
            val = danglingPprTop19.find(i)->second;
            danglingPpr.push_back(val);
        }else danglingPpr.push_back(0.0);
    }

}

void FinalImplementation::initDanglingIndexes() {
    for(int i=0; i<pPpr->V; i++){
        if(pPpr->dangling[i] == 1) pDanglingIndexes.push_back(i);
    }
}

void FinalImplementation::alloc(){
    // Load the input graph and preprocess it;

    pPpr->initialize_graph();
    initDanglingIndexes();
    initDanglingPprTop19();
    initDanglingPpr();
    danglingSize = pDanglingIndexes.size();

    hipMalloc(&x_gpu, sizeof(int) * pPpr->E);
    hipMalloc(&y_gpu, sizeof(int) * pPpr->E);
    hipMalloc(&val_gpu, sizeof(float ) * pPpr->E);


    hipMalloc(&dangling_gpu, sizeof(int) *pPpr->V);
    hipMalloc(&pSquareError_gpu, sizeof(float));

    hipMalloc(&pr_gpu, sizeof(float)*pPpr->V);
    hipMalloc(&pr_old, sizeof(float)*pPpr->V);
    hipMalloc(&pr_gpu_double, sizeof(double)*pPpr->V);

    hipMallocManaged(&pDanglingFact_gpu,sizeof(float)) ;
    hipMalloc(&pDanglingIndexes_gpu, sizeof( int)* danglingSize);

    num_units  = pPpr->E / WARP_SIZE;
    num_warps  = std::min(num_units, WARPS_PER_BLOCK * MAX_BLOCKS);
    num_blocks = (num_warps + (WARPS_PER_BLOCK - 1)) / WARPS_PER_BLOCK;
    num_iters  = (num_units + (num_warps - 1)) / num_warps;
    interval_size = WARP_SIZE * num_iters;
    tail = num_units * WARP_SIZE ;// do the last few nonzeros separately (fewer than WARP_SIZE elements)
    active_warps = (interval_size == 0) ? 0 : (tail + (interval_size - 1)) / interval_size;

    hipMalloc(&temp_rows, sizeof(int) * active_warps);
    hipMalloc(&temp_vals, sizeof(float) * active_warps);

    reducedV = pPpr->V; // 1.500.000 worked well

    BLOCKS_V = (pPpr->V + THREADS - 1)/ THREADS;
    BLOCKS_E = (pPpr->E + THREADS - 1)/ THREADS;
    BLOCKS_ERROR = (reducedV + THREADS - 1)/ THREADS;
    BLOCKS_D = (danglingSize + THREADS - 1)/ THREADS;
    ppVertexConst = (float)(1.0 - pPpr->alpha);
    dampingFract = (float) (pPpr->alpha / pPpr->V);
}

void FinalImplementation::init() {

    valFloat.resize(pPpr->E);
    std::transform(pPpr->val.begin(), pPpr->val.end(), valFloat.begin(), [](double x) { return (float )x;});

    prFloat.resize(pPpr->V);
    std::transform(pPpr->pr.begin(), pPpr->pr.end(), prFloat.begin(), [](double x) { return (float) x; });

    hipMemcpy(x_gpu,pPpr->x.data(), sizeof(int) * pPpr->E, hipMemcpyHostToDevice);
    hipMemcpy(y_gpu,pPpr->y.data(), sizeof(int) * pPpr->E,hipMemcpyHostToDevice);

    hipMemcpy(val_gpu, valFloat.data(), sizeof(float) * pPpr->E, hipMemcpyHostToDevice);
    vectorScalarMul_math<float><<<BLOCKS_E, THREADS>>>(pPpr->alpha, val_gpu, pPpr->E);

    hipMemcpy(dangling_gpu, pPpr->dangling.data(), sizeof(int) * pPpr->V, hipMemcpyHostToDevice);

    hipMemcpy(pDanglingIndexes_gpu, pDanglingIndexes.data(), sizeof(int) * danglingSize, hipMemcpyHostToDevice);

}

void FinalImplementation::reset() {

    if (old_ppr_index != -1) danglingPpr[old_ppr_index] = 0.0;

    if(isDangling(pPpr->personalization_vertex)) {
        danglingPpr[pPpr->personalization_vertex] = 20.0;
        old_ppr_index = pPpr->personalization_vertex;
    }

    init_vector<float><<<BLOCKS_V, THREADS>>>(pr_gpu, pPpr->V, 0.0);
    hipMemcpy(pr_old, prFloat.data(), sizeof(float) * pPpr->V, hipMemcpyHostToDevice);

}

bool FinalImplementation::isDangling(int vertex){
    return (pPpr->dangling[vertex] == 1);
}

void FinalImplementation::execute(int iter) {

    hipStream_t coospmv_stream;
    hipStream_t dangling_stream;
    hipStream_t euclidean_stream;
    hipStream_t init_copy_stream;
    hipStreamCreate(& coospmv_stream);
    hipStreamCreate(&dangling_stream);
    hipStreamCreate(&euclidean_stream);
    hipStreamCreate(&init_copy_stream);

    if(isDangling(pPpr->personalization_vertex)){
        std::copy(danglingPpr.begin(), danglingPpr.end(), pPpr->pr.begin());
        return;
    }
    const int THREADS = 256;
    squareError_cpu = INITIAL_SQUARE_ERROR;
    float heuristic_threshold = 0.0000006;

    for (int i = 0; squareError_cpu > heuristic_threshold && i < pPpr->max_iterations; i++) {

        if (i == 0) init_vector<double><<<BLOCKS_V, THREADS>>>(pr_gpu_double, pPpr->V, 0.0);

        hipMemsetAsync(pDanglingFact_gpu,0.0, sizeof(float), dangling_stream);
        hipMemsetAsync(pSquareError_gpu,0.0, sizeof(float), euclidean_stream);
        init_vector<float><<<BLOCKS_V, THREADS, 0, init_copy_stream>>>(pr_gpu, pPpr->V, 0);

        dangling_kernel< int, float><<<BLOCKS_D, THREADS, THREADS * sizeof(float), dangling_stream>>>(pDanglingIndexes_gpu, pr_old, pDanglingFact_gpu, dampingFract,danglingSize);
        __spmv_coo_flat<int, float, THREADS>(x_gpu, y_gpu, val_gpu, pr_old, pr_gpu, pPpr->E, num_blocks, interval_size, tail,
                                             active_warps, temp_rows, temp_vals);

        hipDeviceSynchronize();

        vectorScalarAddAndIncrement_math<float><<<BLOCKS_V, THREADS>>>(*pDanglingFact_gpu, pr_gpu, pPpr->V, pPpr->personalization_vertex, ppVertexConst);
        hipDeviceSynchronize();
        if((i > 3 && i < 12) || (i > 12 && i % 3 == 0)){
            euclidean_kernel_math < float ><<<BLOCKS_V, THREADS, THREADS * sizeof(float), euclidean_stream>>>(pr_old, pr_gpu, pSquareError_gpu, pPpr->V);
            hipMemcpyAsync(&squareError_cpu, pSquareError_gpu, sizeof(float), hipMemcpyDeviceToHost);
            if(squareError_cpu < heuristic_threshold) squareError_cpu = 0.0;
        }

        copy_vector<float><<<BLOCKS_V, THREADS, 0, init_copy_stream>>>(pr_old,pr_gpu,  pPpr->V);

    }
    hipDeviceSynchronize();
    cast_vector<double, float><<<BLOCKS_V, THREADS>>>(pr_gpu_double,pr_gpu, pPpr->V);

    hipStreamDestroy(coospmv_stream);
    hipStreamDestroy(dangling_stream);
    hipStreamDestroy(euclidean_stream);
    hipStreamDestroy(init_copy_stream);

    //A pointer to the address in base class is used since the validation is done by the base class
    hipMemcpy(pPpr->pr.data(),pr_gpu_double, sizeof (double )*pPpr->V,hipMemcpyDeviceToHost);

}

void FinalImplementation::clean() {
    hipFree(x_gpu);
    hipFree(y_gpu);
    hipFree(val_gpu);

    hipFree(dangling_gpu);
    hipFree(pSquareError_gpu);

    hipFree(pr_gpu);
    hipFree(pr_gpu_double);
    hipFree(pr_old);
    hipFree(pDanglingFact_gpu);

    hipFree(pDanglingIndexes_gpu);

    hipFree(temp_rows);
    hipFree(temp_vals);
}