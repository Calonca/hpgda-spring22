#include "hip/hip_runtime.h"
#include "improvedImplementation.cuh"
#include "../personalized_pagerank.cuh"
#include "../pprFunctions/pprFunctions.cuh"

//convert COO in CSR
void ImprovedImplementation::initCSR() {
    int ptr = 0, previousX;

    if (pPpr->E == 0)
        return;

    previousX = 0;
    xPtr.push_back(0);

    for (int i = 0; i < pPpr->E; i++) {
        while (pPpr->x[i] != previousX) {
            xPtr.push_back(ptr);
            previousX++;
        }
        ptr++;
    }

    for (int i = 0; i < pPpr->V - pPpr->x[pPpr->E - 1]; i++) {
        xPtr.push_back(ptr);
    }
}

void ImprovedImplementation::alloc(){
    // Load the input graph and preprocess it;
    pPpr->initialize_graph();
    initCSR();

    hipMallocManaged(&x_gpu, sizeof(int) * pPpr->E);
    hipMallocManaged(&xPtr_gpu, sizeof(int) * (pPpr->V+1));
    hipMallocManaged(&y_gpu, sizeof(int) * pPpr->E);
    hipMallocManaged(&val_gpu, sizeof(double ) * pPpr->E);


    hipMallocManaged(&dangling_gpu, sizeof(int) *pPpr->V);
    hipMallocManaged(&pSquareError_gpu, sizeof(double));

    hipMallocManaged(&pr_gpu, sizeof(double)*pPpr->V);
    hipMallocManaged(&pr_temp, sizeof(double)*pPpr->V);
    hipMallocManaged(&pr_old, sizeof(double)*pPpr->V);
    hipMallocManaged(&prMinus2, sizeof(double)*pPpr->V);
    hipMallocManaged(&g, sizeof(double)*pPpr->V);
    hipMallocManaged(&h, sizeof(double)*pPpr->V);

    hipMallocManaged(&pDanglingFact_gpu,sizeof(double)) ;
    hipMallocManaged(&count1, sizeof(unsigned int));
    hipMallocManaged(&count2, sizeof(unsigned int));

    BLOCKS_V = (pPpr->V + THREADS - 1)/ THREADS;
    BLOCKS_E = (pPpr->E + THREADS - 1)/ THREADS;
}

void ImprovedImplementation::init() {

}

void ImprovedImplementation::reset() {

    hipMemcpy(x_gpu,pPpr->x.data(), sizeof(int) * pPpr->E, hipMemcpyHostToDevice);

    hipMemcpy(xPtr_gpu,xPtr.data(), sizeof(int) * (pPpr->V + 1), hipMemcpyHostToDevice);

    hipMemcpy(y_gpu,pPpr->y.data(), sizeof(int) * pPpr->E,hipMemcpyHostToDevice);
    hipMemcpy(val_gpu,pPpr->val.data(), sizeof(double ) * pPpr->E,hipMemcpyHostToDevice);
    vectorScalarMul<double><<<BLOCKS_E, THREADS>>>(pPpr->alpha, val_gpu, pPpr->E);

    hipMemcpy(dangling_gpu, pPpr->dangling.data(), sizeof(int) * pPpr->V, hipMemcpyHostToDevice);

    hipMemset(pr_gpu,0.0, sizeof(double)*pPpr->V);
    hipMemset(pr_temp,0.0, sizeof(double)*pPpr->V);
    hipMemcpy(pr_old,pPpr->pr.data() ,sizeof(double)*pPpr->V,hipMemcpyHostToDevice);



}

void ImprovedImplementation::execute(int iter) {
    double squareError_cpu = INITIAL_SQUARE_ERROR;
    int aitken = 0;

    double dampingFract = (double) pPpr->alpha / pPpr->V;

    for (int i = 0; squareError_cpu > pPpr->convergence_threshold && i < pPpr->max_iterations; i++) {

        init_vector<double><<<BLOCKS_V, THREADS>>>(pr_gpu, pPpr->V, 0);
        init_vector<double><<<BLOCKS_V, THREADS>>>(pDanglingFact_gpu, 1, 0);
        init_vector<double><<<BLOCKS_V, THREADS>>>(pSquareError_gpu, 1, 0);

        hipDeviceSynchronize();

        if (!aitken) {

            dot_product_kernel<int, double><<<BLOCKS_V, THREADS, THREADS * sizeof(double)>>>(dangling_gpu, pr_old, pDanglingFact_gpu, dampingFract, pPpr->V);
            cooSPMV<int, double><<<BLOCKS_E, THREADS>>>(x_gpu, y_gpu, val_gpu, pPpr->E, pr_old, pr_gpu); // needs improvement!!
            hipDeviceSynchronize();

            vectorScalarAddAndIncrement<double><<<BLOCKS_V, THREADS>>>(*pDanglingFact_gpu, pr_gpu,
                                                                       pPpr->V, pPpr->personalization_vertex,
                                                                       1.0 - pPpr->alpha);
            hipDeviceSynchronize();

            compute_square_error_gpu<double><<<BLOCKS_V, THREADS, THREADS * sizeof(double)>>>(pr_old, pr_gpu, pSquareError_gpu, pPpr->V);

            hipMemcpy(&squareError_cpu, pSquareError_gpu, sizeof(double), hipMemcpyDeviceToHost);
            squareError_cpu = std::sqrt(squareError_cpu);


            if (i == 30) hipMemcpy(prMinus2, pr_old, sizeof(double) * pPpr->V, hipMemcpyDeviceToDevice);


            if (i==30)  aitken = 1;

        }

        if(aitken){
            compute_aikten_x_math<<<BLOCKS_V, THREADS>>>(pr_gpu, pr_old, prMinus2, pPpr->V);
            aitken = 0;
            hipDeviceSynchronize();
        }

        hipMemcpy(pr_old, pr_gpu, sizeof(double) * pPpr->V, hipMemcpyDeviceToDevice);

    }

    hipDeviceSynchronize();

    //A pointer to the address in base class is used since the validation is done by the base class
    hipMemcpy(pPpr->pr.data(),pr_gpu,sizeof (double )*pPpr->V,hipMemcpyDeviceToHost);
}

void ImprovedImplementation::clean() {
    hipFree(x_gpu);
    hipFree(y_gpu);
    hipFree(val_gpu);
    hipFree(dangling_gpu);
    hipFree(pSquareError_gpu);
    hipFree(pr_gpu);
    hipFree(pr_temp);
    hipFree(pr_old);
    hipFree(pDanglingFact_gpu);
    hipFree(count1);
    hipFree(count2);
}
